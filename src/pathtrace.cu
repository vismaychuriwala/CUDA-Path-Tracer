#include "hip/hip_runtime.h"
#include "pathtrace.h"

#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/remove.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/tuple.h>
#include <thrust/gather.h>

#include "sceneStructs.h"
#include "scene.h"
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"
#include "utilities.h"
#include "intersections.h"
#include "interactions.h"

#define ERRORCHECK 1
const bool COALESCED = 1;

#define FILENAME (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
#define checkCUDAError(msg) checkCUDAErrorFn(msg, FILENAME, __LINE__)
void checkCUDAErrorFn(const char* msg, const char* file, int line)
{
#if ERRORCHECK
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (hipSuccess == err)
    {
        return;
    }

    fprintf(stderr, "CUDA error");
    if (file)
    {
        fprintf(stderr, " (%s:%d)", file, line);
    }
    fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
#ifdef _WIN32
    getchar();
#endif // _WIN32
    exit(EXIT_FAILURE);
#endif // ERRORCHECK
}

__host__ __device__
thrust::default_random_engine makeSeededRandomEngine(int iter, int index, int depth)
{
    int h = utilhash((1 << 31) | (depth << 22) | iter) ^ utilhash(index);
    return thrust::default_random_engine(h);
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* pbo, glm::ivec2 resolution, int iter, glm::vec3* image)
{
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < resolution.x && y < resolution.y)
    {
        int index = x + (y * resolution.x);
        glm::vec3 pix = image[index];

        glm::ivec3 color;
        color.x = glm::clamp((int)(pix.x / iter * 255.0), 0, 255);
        color.y = glm::clamp((int)(pix.y / iter * 255.0), 0, 255);
        color.z = glm::clamp((int)(pix.z / iter * 255.0), 0, 255);

        // Each thread writes one pixel location in the texture (textel)
        pbo[index].w = 0;
        pbo[index].x = color.x;
        pbo[index].y = color.y;
        pbo[index].z = color.z;
    }
}

static Scene* hst_scene = NULL;
static GuiDataContainer* guiData = NULL;
static glm::vec3* dev_image = NULL;
static Geom* dev_geoms = NULL;
static Material* dev_materials = NULL;
static PathSegment* dev_paths = NULL;
static ShadeableIntersection* dev_intersections = NULL;

int* dev_keys = nullptr;
int* dev_indices = nullptr;
ShadeableIntersection* dev_inter_tmp;
PathSegment* dev_paths_tmp;

// TODO: static variables for device memory, any extra info you need, etc
// ...

void InitDataContainer(GuiDataContainer* imGuiData)
{
    guiData = imGuiData;
}

void pathtraceInit(Scene* scene)
{
    hst_scene = scene;

    const Camera& cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    hipMalloc(&dev_image, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));

    hipMalloc(&dev_paths, pixelcount * sizeof(PathSegment));

    hipMalloc(&dev_geoms, scene->geoms.size() * sizeof(Geom));
    hipMemcpy(dev_geoms, scene->geoms.data(), scene->geoms.size() * sizeof(Geom), hipMemcpyHostToDevice);

    hipMalloc(&dev_materials, scene->materials.size() * sizeof(Material));
    hipMemcpy(dev_materials, scene->materials.data(), scene->materials.size() * sizeof(Material), hipMemcpyHostToDevice);

    hipMalloc(&dev_intersections, pixelcount * sizeof(ShadeableIntersection));
    hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

    // TODO: initialize any extra device memeory you need

    if (COALESCED) {
        hipMalloc(&dev_keys, pixelcount * sizeof(int));
        hipMalloc(&dev_indices, pixelcount * sizeof(int));
        hipMalloc(&dev_inter_tmp, pixelcount * sizeof(ShadeableIntersection));
        hipMalloc(&dev_paths_tmp, pixelcount * sizeof(PathSegment));
    }
    checkCUDAError("pathtraceInit");
}

void pathtraceFree()
{
    hipFree(dev_image);  // no-op if dev_image is null
    hipFree(dev_paths);
    hipFree(dev_geoms);
    hipFree(dev_materials);
    hipFree(dev_intersections);

    // TODO: clean up any extra device memory you created
    if (COALESCED) {
        hipFree(dev_keys);
        hipFree(dev_indices);
        hipFree(dev_inter_tmp);
        hipFree(dev_paths_tmp);
    }

    checkCUDAError("pathtraceFree");
}

__device__ glm::vec2 concentricSampleDisk(float u1, float u2) {
    float sx = 2.0f * u1 - 1.0f;
    float sy = 2.0f * u2 - 1.0f;

    if (sx == 0.0f && sy == 0.0f) return glm::vec2(0.0f);

    float r, theta;
    if (fabsf(sx) > fabsf(sy)) {
        r = sx;
        theta = (PI / 4) * (sy / sx);
    } else {
        r = sy;
        theta = (PI / 2) - (PI / 4) * (sx / sy);
    }
    return glm::vec2(r * cosf(theta), r * sinf(theta));
}

/**
* Generate PathSegments with rays from the camera through the screen into the
* scene, which is the first bounce of rays.
*
* Antialiasing - add rays for sub-pixel sampling
* motion blur - jitter rays "in time"
* lens effect - jitter ray origin positions based on a lens
*/
__global__ void generateRayFromCamera(Camera cam, int iter, int traceDepth, PathSegment* pathSegments)
{
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < cam.resolution.x && y < cam.resolution.y) {
        int index = x + (y * cam.resolution.x);
        PathSegment& segment = pathSegments[index];

        segment.ray.origin = cam.position;
        segment.color = glm::vec3(1.0f, 1.0f, 1.0f);

        // TODO: implement antialiasing by jittering the ray
        thrust::default_random_engine rng = makeSeededRandomEngine(iter, index, 0);
        thrust::normal_distribution<float> normal(0.0f, 0.005f);
        float jitterX = normal(rng);
        float jitterY = normal(rng);
        jitterX = fminf(fmaxf(jitterX, -0.5f), 0.5f);
        jitterY = fminf(fmaxf(jitterY, -0.5f), 0.5f);
        float px = (float)x + jitterX;
        float py = (float)y + jitterY;
        glm::vec3 dir_pinhole = glm::normalize(cam.view
            - cam.right * cam.pixelLength.x * (px - (float)cam.resolution.x * 0.5f)
            - cam.up * cam.pixelLength.y * (py - (float)cam.resolution.y * 0.5f)
        );


        glm::vec3 rayOrigin = cam.position;
        glm::vec3 rayDir = dir_pinhole;

        if (cam.lensRadius > 0.0f) {
            // Uniform randoms using your thrust RNG
            thrust::uniform_real_distribution<float> uni01(0.0f, 1.0f);
            float r1 = uni01(rng);
            float r2 = uni01(rng);

            // compute focal point along the pinhole ray: find t so that camera + t*dir_pinhole lies
            // on plane at distance focalDistance along cam.view.
            // Assumes cam.view is normalized.
            float denom = glm::dot(dir_pinhole, cam.view);
            // avoid /0
            denom = (fabsf(denom) < 1e-6f) ? 1e-6f * (denom >= 0.0f ? 1.0f : -1.0f) : denom;
            float t_focus = cam.focalDistance / denom;
            glm::vec3 p_focus = cam.position + dir_pinhole * t_focus;

            // sample lens disk and offset origin
            glm::vec2 lensSample = concentricSampleDisk(r1, r2) * cam.lensRadius;
            rayOrigin = cam.position + cam.right * lensSample.x + cam.up * lensSample.y;

            // new direction goes from sampled lens point to focal point
            rayDir = glm::normalize(p_focus - rayOrigin);
        }
        segment.ray.origin = rayOrigin;
        segment.ray.direction = rayDir;
        segment.pixelIndex = index;
        segment.remainingBounces = traceDepth;
    }
}

// TODO:
// computeIntersections handles generating ray intersections ONLY.
// Generating new rays is handled in your shader(s).
// Feel free to modify the code below.
__global__ void computeIntersections(
    int depth,
    int num_paths,
    PathSegment* pathSegments,
    Geom* geoms,
    int geoms_size,
    ShadeableIntersection* intersections)
{
    int path_index = blockIdx.x * blockDim.x + threadIdx.x;

    if (path_index < num_paths)
    {
        PathSegment pathSegment = pathSegments[path_index];

        float t;
        glm::vec3 intersect_point;
        glm::vec3 normal;
        float t_min = FLT_MAX;
        int hit_geom_index = -1;
        bool outside = true;

        glm::vec3 tmp_intersect;
        glm::vec3 tmp_normal;

        // naive parse through global geoms

        for (int i = 0; i < geoms_size; i++)
        {
            Geom& geom = geoms[i];

            if (geom.type == CUBE)
            {
                t = boxIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
            }
            else if (geom.type == SPHERE)
            {
                t = sphereIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
            }
            // TODO: add more intersection tests here... triangle? metaball? CSG?

            // Compute the minimum t from the intersection tests to determine what
            // scene geometry object was hit first.
            if (t > 0.0f && t_min > t)
            {
                t_min = t;
                hit_geom_index = i;
                intersect_point = tmp_intersect;
                normal = tmp_normal;
            }
        }

        if (hit_geom_index == -1)
        {
            intersections[path_index].t = -1.0f;
        }
        else
        {
            // The ray hits something
            intersections[path_index].t = t_min;
            intersections[path_index].materialId = geoms[hit_geom_index].materialid;
            intersections[path_index].surfaceNormal = normal;
        }
    }
}

__global__ void shadeRealMaterial(
    int iter,
    int num_paths,
    ShadeableIntersection* shadeableIntersections,
    PathSegment* pathSegments,
    Material* materials)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_paths)
    {
        ShadeableIntersection intersection = shadeableIntersections[idx];
        if (intersection.t > 0.0f) // if the intersection exists...
        {
          // Set up the RNG
            thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, 0);
            thrust::uniform_real_distribution<float> u01(0, 1);

            Material material = materials[intersection.materialId];
            glm::vec3 materialColor = material.color;
            PathSegment &pathSegment = pathSegments[idx];

            // If the material indicates that the object was a light, "light" the ray
            if (material.emittance > 0.0f) {
                pathSegment.color *= (materialColor * material.emittance);
                pathSegment.remainingBounces = 0;
            }
            // Otherwise, do some pseudo-lighting computation. This is actually more
            // like what you would expect from shading in a rasterizer like OpenGL.
            // TODO: replace this! you should be able to start with basically a one-liner
            else {
                if (pathSegment.remainingBounces > 0) {
                    glm::vec3 hitPoint = pathSegment.ray.origin + intersection.t * pathSegment.ray.direction;
                    scatterRay(pathSegment, hitPoint, intersection.surfaceNormal, material, rng);
                }
                // float lightTerm = glm::dot(intersection.surfaceNormal, glm::vec3(0.0f, 1.0f, 0.0f));
                // pathSegments[idx].color *= (materialColor * lightTerm) * 0.3f + ((1.0f - intersection.t * 0.02f) * materialColor) * 0.7f;
                // pathSegments[idx].color *= u01(rng); // apply some noise because why not
            }
            // If there was no intersection, color the ray black.
            // Lots of renderers use 4 channel color, RGBA, where A = alpha, often
            // used for opacity, in which case they can indicate "no opacity".
            // This can be useful for post-processing and image compositing.
        }
        else {
            pathSegments[idx].color = glm::vec3(0.0f);
            pathSegments[idx].remainingBounces = 0;
        }
    }
}

// Add the current iteration's output to the overall image
__global__ void gatherImage(int nPaths, glm::vec3* image, PathSegment* iterationPaths)
{
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (index < nPaths)
    {
        PathSegment iterationPath = iterationPaths[index];
        if (iterationPath.remainingBounces == 0) {
            image[iterationPath.pixelIndex] += iterationPath.color;
        }
    }
}

// Set the sorting keys to materialId
__global__ void kernSetKeys(int nPaths, int* keys, const ShadeableIntersection* intersections)
{
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index < nPaths)
    {
        keys[index] = intersections[index].materialId;
    }
}

struct PathIsDead {
  __host__ __device__
  bool operator()(const PathSegment &p) const {
    return p.remainingBounces == 0;
  }
};

int compactPaths_inplace(PathSegment* d_paths, int num_paths) {
  thrust::device_ptr<PathSegment> dev_ptr(d_paths);
  auto new_end = thrust::remove_if(dev_ptr, dev_ptr + num_paths, PathIsDead());
  int new_num_paths = static_cast<int>(new_end - dev_ptr);
  return new_num_paths;
}

// Sort Intersections and Paths according to index
__global__ void kernGatherArrays(int nPaths, int* indices, ShadeableIntersection* inter_out, const ShadeableIntersection* inter_in,
PathSegment* path_out, const PathSegment* path_in
)
{
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index < nPaths)
    {
        int src = indices[index];
        inter_out[index] = inter_in[src];
        path_out[index] = path_in[src];
    }
}

/**
 * Wrapper for the __global__ call that sets up the kernel calls and does a ton
 * of memory management
 */
void pathtrace(uchar4* pbo, int frame, int iter)
{
    const int traceDepth = hst_scene->state.traceDepth;
    const Camera& cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    // 2D block for generating ray from camera
    const dim3 blockSize2d(8, 8);
    const dim3 blocksPerGrid2d(
        (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
        (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

    // 1D block for path tracing
    const int blockSize1d = 128;

    ///////////////////////////////////////////////////////////////////////////

    // Recap:
    // * Initialize array of path rays (using rays that come out of the camera)
    //   * You can pass the Camera object to that kernel.
    //   * Each path ray must carry at minimum a (ray, color) pair,
    //   * where color starts as the multiplicative identity, white = (1, 1, 1).
    //   * This has already been done for you.
    // * For each depth:
    //   * Compute an intersection in the scene for each path ray.
    //     A very naive version of this has been implemented for you, but feel
    //     free to add more primitives and/or a better algorithm.
    //     Currently, intersection distance is recorded as a parametric distance,
    //     t, or a "distance along the ray." t = -1.0 indicates no intersection.
    //     * Color is attenuated (multiplied) by reflections off of any object
    //   * TODO: Stream compact away all of the terminated paths.
    //     You may use either your implementation or `thrust::remove_if` or its
    //     cousins.
    //     * Note that you can't really use a 2D kernel launch any more - switch
    //       to 1D.
    //   * TODO: Shade the rays that intersected something or didn't bottom out.
    //     That is, color the ray by performing a color computation according
    //     to the shader, then generate a new ray to continue the ray path.
    //     We recommend just updating the ray's PathSegment in place.
    //     Note that this step may come before or after stream compaction,
    //     since some shaders you write may also cause a path to terminate.
    // * Finally, add this iteration's results to the image. This has been done
    //   for you.

    // TODO: perform one iteration of path tracing

    generateRayFromCamera<<<blocksPerGrid2d, blockSize2d>>>(cam, iter, traceDepth, dev_paths);
    checkCUDAError("generate camera ray");

    int depth = 0;
    PathSegment* dev_path_end = dev_paths + pixelcount;
    int num_paths = dev_path_end - dev_paths;
    // --- PathSegment Tracing Stage ---
    // Shoot ray into scene, bounce between objects, push shading chunks

    bool iterationComplete = false;
    while (!iterationComplete)
    {
        // clean shading chunks
        hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

        // tracing
        dim3 numblocksPathSegmentTracing = (num_paths + blockSize1d - 1) / blockSize1d;
        computeIntersections<<<numblocksPathSegmentTracing, blockSize1d>>> (
            depth,
            num_paths,
            dev_paths,
            dev_geoms,
            hst_scene->geoms.size(),
            dev_intersections
        );
        checkCUDAError("trace one bounce");
        hipDeviceSynchronize();
        depth++;

        // TODO:
        // --- Shading Stage ---
        // Shade path segments based on intersections and generate new rays by
        // evaluating the BSDF.
        // Start off with just a big kernel that handles all the different
        // materials you have in the scenefile.
        // TODO: compare between directly shading the path segments and shading
        // path segments that have been reshuffled to be contiguous in memory.

        if (COALESCED) {

            thrust::device_ptr<int> d_keys(dev_keys);
        
            kernSetKeys<<<numblocksPathSegmentTracing, blockSize1d>>> (num_paths, dev_keys, dev_intersections);
            thrust::device_ptr<int> d_idx(dev_indices);
            thrust::sequence(thrust::device, d_idx, d_idx + num_paths);
            thrust::sort_by_key(thrust::device, d_keys, d_keys + num_paths, d_idx);
            kernGatherArrays<<<numblocksPathSegmentTracing, blockSize1d>>>(num_paths, dev_indices, dev_inter_tmp, dev_intersections, dev_paths_tmp, dev_paths);
            std::swap(dev_inter_tmp, dev_intersections);
            std::swap(dev_paths_tmp, dev_paths);
        }

        shadeRealMaterial<<<numblocksPathSegmentTracing, blockSize1d>>>(
            iter,
            num_paths,
            dev_intersections,
            dev_paths,
            dev_materials
        );
        hipDeviceSynchronize();

        int n = num_paths;
        int blocksGather = (n + blockSize1d - 1) / blockSize1d;
        gatherImage << <blocksGather, blockSize1d >> > (n, dev_image, dev_paths);
        hipDeviceSynchronize();
        num_paths = compactPaths_inplace(dev_paths, num_paths);
        if (num_paths == 0) {
            iterationComplete = true;
        }
        if (guiData != NULL)
        {
            guiData->TracedDepth = depth;
        }
    }

    //// Assemble this iteration and apply it to the image
    //dim3 numBlocksPixels = (pixelcount + blockSize1d - 1) / blockSize1d;
    //finalGather<<<numBlocksPixels, blockSize1d>>>(num_paths, dev_image, dev_paths);

    ///////////////////////////////////////////////////////////////////////////

    // Send results to OpenGL buffer for rendering
    sendImageToPBO<<<blocksPerGrid2d, blockSize2d>>>(pbo, cam.resolution, iter, dev_image);

    // Retrieve image from GPU
    hipMemcpy(hst_scene->state.image.data(), dev_image,
        pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);

    checkCUDAError("pathtrace");
}
